#include "hip/hip_runtime.h"


#include <chrono>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsparse.h>

#include <hipcub/hipcub.hpp>

#include "gasp.h"
#include "test_common.h"


using namespace testing;
using namespace gasp;

template <typename SR, typename D, typename I>
void check_correctness(const size_t m,
                        const size_t n,
                        const size_t nnz,
                        hipsparseSpMatDescr_t& A,
                        hipsparseDnVecDescr_t& x, 
                        hipsparseDnVecDescr_t& y, 
                        GaspCsr<D,I>& gasp_A,
                        D * d_x, D * d_y)
{
    hipsparseHandle_t cusparseHandle;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    const float alpha = 1.0;
    const float beta = 0.0;

    size_t buf_size;
    void * buf;

    CUSPARSE_CHECK(hipsparseSpMV_bufferSize(cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, A, x,
                                            &beta, y,
                                            HIP_R_32F,
                                            HIPSPARSE_SPMV_ALG_DEFAULT,
                                            &buf_size));

    CUDA_CHECK(hipMalloc(&buf, buf_size));
    CUSPARSE_CHECK(hipsparseSpMV(cusparseHandle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, A, x,
                                &beta, y,
                                HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT,
                                buf));

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(buf));
    D * h_correct = new D[m];
    CUDA_CHECK(hipMemcpy(h_correct, d_y, sizeof(D)*m, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_y, 0, sizeof(D)*m));

    SpMV_host<PlusTimesSemiring<float>, float, uint32_t>(gasp_A, d_x, d_y);
    CUDA_CHECK(hipDeviceSynchronize());

    D * h_computed = new D[m];
    CUDA_CHECK(hipMemcpy(h_computed, d_y, sizeof(D)*m, hipMemcpyDeviceToHost));

    for (int i=0; i<m; i++) {
        assert(fabs((h_computed[i] - h_correct[i])) < EPS);
    }
    std::cout<<BRIGHT_GREEN<<"Correctness for SpMV passed!"<<RESET<<std::endl;

    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));

    delete[] h_correct;
    delete[] h_computed;
}

int main(int argc, char ** argv)
{
    hipsparseHandle_t cusparseHandle;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    const size_t m = std::atol(argv[1]);
    const size_t n = std::atol(argv[2]);
    const size_t nnz = std::atol(argv[3]);
    const size_t n_iters = std::atoi(argv[4]);
    const std::string action(argv[5]);

    float * d_vals, * d_x, * d_y;
    uint32_t * d_colinds, * d_rowptrs;

    init_sparse_mat_csr<float, RandomCsrInitializer>
                        (m, n, nnz, 
                        &d_vals, &d_colinds, &d_rowptrs, 
                        RandomCsrInitializer());
    init_dense_vec(n, &d_x);
    init_dense_vec(m, &d_y);

    hipsparseDnVecDescr_t x;
    hipsparseDnVecDescr_t y;
    hipsparseSpMatDescr_t A;

    CUSPARSE_CHECK(hipsparseCreateDnVec(&x, n, d_x, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&y, m, d_y, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateCsr(&A,
                                     m, n, nnz,
                                     d_rowptrs,
                                     d_colinds,
                                     d_vals,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_32F));

    GaspCsr gasp_A(m, n, nnz,
                   d_vals, d_colinds, d_rowptrs);

    if (action.compare("correctness")==0) { 
        check_correctness<PlusTimesSemiring<float>, float, uint32_t>
            (m, n, nnz, A, x, y, gasp_A, d_x, d_y);
    } else if (action.compare("benchmark")==0) {
        const float alpha = 1.0;
        const float beta = 0.0;

        size_t buf_size;
        void * buf;

        const char * label_cusparse = "SpMV_cusparse";

        start_timer(label_cusparse);
        for (int i=0; i<n_iters; i++) {

            /* BENCHMARK CUSPARSE */
            CUSPARSE_CHECK(hipsparseSpMV_bufferSize(cusparseHandle,
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, A, x,
                                                    &beta, y,
                                                    HIP_R_32F,
                                                    HIPSPARSE_SPMV_ALG_DEFAULT,
                                                    &buf_size));

            CUDA_CHECK(hipMalloc(&buf, buf_size));
            CUSPARSE_CHECK(hipsparseSpMV(cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, A, x,
                                        &beta, y,
                                        HIP_R_32F,
                                        HIPSPARSE_SPMV_ALG_DEFAULT,
                                        buf));

            CUDA_CHECK(hipDeviceSynchronize());

            CUDA_CHECK(hipFree(buf));
        }

        end_timer(label_cusparse);
        measure_gflops(label_cusparse, 2*nnz*n_iters);

        print_time(label_cusparse);
        print_gflops(label_cusparse);


        const char * label_gasp = "SpMV_gasp";

        start_timer(label_gasp);
        for (int i=0; i<n_iters; i++) {
            SpMV_host<PlusTimesSemiring<float>, float, uint32_t>(gasp_A, d_x, d_y);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        end_timer(label_gasp);
        measure_gflops(label_gasp, 2*nnz*n_iters);

        print_time(label_gasp);
        print_gflops(label_gasp);
    }

    CUSPARSE_CHECK(hipsparseDestroySpMat(A));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(x));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(y));

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));


    return 0;
}
