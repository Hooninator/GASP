#include "hip/hip_runtime.h"


#include <chrono>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsparse.h>

#include <hipcub/hipcub.hpp>

#include "gasp.h"
#include "test_common.h"


using namespace testing;
using namespace gasp;

template <typename SR, typename Matrix, typename D>
void check_correctness(const size_t m,
                        const size_t n,
                        const size_t nnz,
                        hipsparseSpMatDescr_t& A,
                        hipsparseDnVecDescr_t& x, 
                        hipsparseDnVecDescr_t& y, 
                        Matrix & gasp_A,
                        D * d_x, D * d_y)
{
    hipsparseHandle_t cusparseHandle;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    const float alpha = 1.0;
    const float beta = 0.0;

    size_t buf_size;
    void * buf;

    CUSPARSE_CHECK(hipsparseSpMV_bufferSize(cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, A, x,
                                            &beta, y,
                                            HIP_R_32F,
                                            HIPSPARSE_SPMV_CSR_ALG1,
                                            &buf_size));

    CUDA_CHECK(hipMalloc(&buf, buf_size));
    CUSPARSE_CHECK(hipsparseSpMV(cusparseHandle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, A, x,
                                &beta, y,
                                HIP_R_32F,
                                HIPSPARSE_SPMV_CSR_ALG1,
                                buf));

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(buf));
    D * h_correct = new D[m];
    CUDA_CHECK(hipMemcpy(h_correct, d_y, sizeof(D)*m, hipMemcpyDeviceToHost));



    CUDA_CHECK(hipMemset(d_y, 0, sizeof(D)*m));

    SpMV_host<SpMVWarp<PlusTimesSemiring<float>>>(gasp_A, d_x, d_y);
    CUDA_CHECK(hipDeviceSynchronize());

    D * h_computed = new D[m];
    CUDA_CHECK(hipMemcpy(h_computed, d_y, sizeof(D)*m, hipMemcpyDeviceToHost));

    for (size_t i=0; i<m; i++) {
        assert(fabs((h_computed[i] - h_correct[i])) < EPS);
    }
    std::cout<<BRIGHT_GREEN<<"Correctness for warp SpMV passed!"<<RESET<<std::endl;



    CUDA_CHECK(hipMemset(d_y, 0, sizeof(D)*m));

    SpMV_host<SpMVScalar<PlusTimesSemiring<float>>>(gasp_A, d_x, d_y);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_computed, d_y, sizeof(D)*m, hipMemcpyDeviceToHost));

    for (size_t i=0; i<m; i++) {
        assert(fabs((h_computed[i] - h_correct[i])) < EPS);
    }
    std::cout<<BRIGHT_GREEN<<"Correctness for scalar SpMV passed!"<<RESET<<std::endl;

    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));

    delete[] h_correct;
    delete[] h_computed;
}

int main(int argc, char ** argv)
{
    hipsparseHandle_t cusparseHandle;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    const size_t m = std::atol(argv[1]);
    const size_t n = std::atol(argv[2]);
    const size_t nnz = std::atol(argv[3]);
    const size_t n_iters = std::atoi(argv[4]);
    const std::string action(argv[5]);

    float * d_vals, * d_x, * d_y;
    int64_t * d_colinds, * d_rowptrs;

    init_sparse_mat_csr<float, RandomCsrInitializer>
                        (m, n, nnz, 
                        &d_vals, &d_colinds, &d_rowptrs, 
                        RandomCsrInitializer());
    init_dense_vec(n, &d_x);
    init_dense_vec(m, &d_y);

    hipsparseDnVecDescr_t x;
    hipsparseDnVecDescr_t y;
    hipsparseSpMatDescr_t A;

    CUSPARSE_CHECK(hipsparseCreateDnVec(&x, n, d_x, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&y, m, d_y, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateCsr(&A,
                                     m, n, nnz,
                                     d_rowptrs,
                                     d_colinds,
                                     d_vals,
                                     HIPSPARSE_INDEX_64I,
                                     HIPSPARSE_INDEX_64I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_32F));

    GaspCsr gasp_A(m, n, nnz, d_vals, d_colinds, d_rowptrs);

    if (action.compare("correctness")==0) { 
        check_correctness<PlusTimesSemiring<float>>
            (m, n, nnz, A, x, y, gasp_A, d_x, d_y);
    } else if (action.compare("benchmark")==0) {
        const float alpha = 1.0;
        const float beta = 0.0;

        size_t buf_size;
        void * buf;

        const char * label_cusparse = "SpMV_cusparse";

        start_timer(label_cusparse);

        /* BENCHMARK CUSPARSE */
        for (int i=0; i<n_iters; i++) {

            CUSPARSE_CHECK(hipsparseSpMV_bufferSize(cusparseHandle,
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, A, x,
                                                    &beta, y,
                                                    HIP_R_32F,
                                                    HIPSPARSE_SPMV_CSR_ALG1,
                                                    &buf_size));

            CUDA_CHECK(hipMalloc(&buf, buf_size));
            CUSPARSE_CHECK(hipsparseSpMV(cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, A, x,
                                        &beta, y,
                                        HIP_R_32F,
                                        HIPSPARSE_SPMV_CSR_ALG1,
                                        buf));

            CUDA_CHECK(hipDeviceSynchronize());

            CUDA_CHECK(hipFree(buf));
        }

        end_timer(label_cusparse);
        measure_gflops(label_cusparse, 2*nnz*n_iters);

        print_time(label_cusparse);
        print_gflops(label_cusparse);


        const char * label_gasp_warp = "SpMV_gasp_warp";

        start_timer(label_gasp_warp);

        for (int i=0; i<n_iters; i++) {
            SpMV_host<SpMVWarp<PlusTimesSemiring<float>>>(gasp_A, d_x, d_y);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        end_timer(label_gasp_warp);
        measure_gflops(label_gasp_warp, 2*nnz*n_iters);

        print_time(label_gasp_warp);
        print_gflops(label_gasp_warp);

        const char * label_gasp_scalar = "SpMV_gasp_scalar";

        start_timer(label_gasp_scalar);

        for (int i=0; i<n_iters; i++) {
            SpMV_host<SpMVScalar<PlusTimesSemiring<float>>>(gasp_A, d_x, d_y);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        end_timer(label_gasp_scalar);
        measure_gflops(label_gasp_scalar, 2*nnz*n_iters);

        print_time(label_gasp_scalar);
        print_gflops(label_gasp_scalar);
    }

    CUSPARSE_CHECK(hipsparseDestroySpMat(A));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(x));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(y));

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));


    return 0;
}
