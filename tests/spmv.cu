

#include <chrono>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsparse.h>

#include <hipcub/hipcub.hpp>

#include "test_common.h"

using namespace testing;


int main(int argc, char ** argv)
{
    hipsparseHandle_t cusparseHandle;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    size_t m = std::atol(argv[1]);
    size_t n = std::atol(argv[2]);
    size_t nnz = std::atol(argv[3]);
    size_t n_iters = std::atoi(argv[4]);

    float * d_vals, * d_vec, * d_result;
    uint32_t * d_colinds, * d_rowptrs;

    init_sparse_mat_csr<float, RandomCsrInitializer>
                        (m, n, nnz, 
                        &d_vals, &d_colinds, &d_rowptrs, 
                        RandomCsrInitializer());
    init_dense_vec(n, &d_vec);
    init_dense_vec(m, &d_result);

    hipsparseDnVecDescr_t x;
    hipsparseDnVecDescr_t y;
    hipsparseSpMatDescr_t A;

    CUSPARSE_CHECK(hipsparseCreateDnVec(&x, n, d_vec, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&y, m, d_result, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateCsr(&A,
                                     m, n, nnz,
                                     d_rowptrs,
                                     d_colinds,
                                     d_vals,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_32F));

    float alpha = 1.0;
    float beta = 0.0;

    size_t buf_size;
    void * buf;

    const char * label_cusparse = "SpMV_cusparse";

    start_timer(label_cusparse);
    for (int i=0; i<n_iters; i++) {

        /* BENCHMARK CUSPARSE */
        CUSPARSE_CHECK(hipsparseSpMV_bufferSize(cusparseHandle,
                                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                &alpha, A, x,
                                                &beta, y,
                                                HIP_R_32F,
                                                HIPSPARSE_SPMV_ALG_DEFAULT,
                                                &buf_size));

        CUDA_CHECK(hipMalloc(&buf, buf_size));
        CUSPARSE_CHECK(hipsparseSpMV(cusparseHandle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, A, x,
                                    &beta, y,
                                    HIP_R_32F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT,
                                    buf));



        CUDA_CHECK(hipFree(buf));
    }

    end_timer(label_cusparse);
    measure_gflops(label_cusparse, 2*nnz*n_iters);

    print_time(label_cusparse);
    print_gflops(label_cusparse);

    CUSPARSE_CHECK(hipsparseDestroySpMat(A));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(x));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(y));


    CUDA_CHECK(hipFree(d_vals));
    CUDA_CHECK(hipFree(d_colinds));
    CUDA_CHECK(hipFree(d_rowptrs));

    CUDA_CHECK(hipFree(d_vec));
    CUDA_CHECK(hipFree(d_result));

    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));


    return 0;
}
